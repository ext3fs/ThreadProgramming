#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
using namespace std;

const int kcnt = 2e9;

__global__ void add(int, float* x, float* y, float* z)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	
	if(i < n)
		z[i] = x[i] + y[i];
}

int main(void)	
{
	vector<int> x(kcnt, 1);
	vector<int> y(kcnt, 2);
	vector<int> z(kcnt);
	long long sum = 0;

	float *divice_x, *divice_y, *divice_z;
	hipMalloc(&divece_x, kcnt * sizeof(float));
	hipMalloc(&divece_y, kcnt * sizeof(float));
	hipMalloc(&divece_z, kcnt * sizeof(float));

	hipMemcpy(divice_x, x.data(), kcnt*sizeof(float), cudaMemcpyHostToDivice);
	hipMemcpy(divice_y, y.data(), kcnt*sizeof(float), cudaMemcpyHostToDivice);

	add<<<1,kcnt>>>(divice_x, divice_y, divice_z, kcnt);
	hipMemcpy(z.data(), divice_z, kcnt*sizeof(float), cudaMemcpyDivideToHost);

	cudafree(divice_x);
	cudafree(divice_y);
	cudafree(divice_z);

	for(int i=0; i<kcnt; ++i)
		sum += z[i];		
	
	cout << sum << endl;	
	return 0;
}

